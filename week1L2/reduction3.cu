#include<hip/hip_runtime.h>
#include<stdio.h>


#define ELEMENT_PER_BLOCK 256

__global__ void reductionKernel(int *d_input, int *d_output, int N) {
    __shared__ int sdata[ELEMENT_PER_BLOCK];
    int tid = threadIdx.x;
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    sdata[tid] = (i < N) ? d_input[i] : 0;
    __syncthreads();

    // Perform reduction in shared memory
    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s) {
            sdata[tid] += sdata[tid + s];
        }
        __syncthreads();
    }
    // Write result for this block to global memory
    if (tid == 0) {
        atomicAdd(d_output, sdata[0]);
    }
}

int main() {
    int N = 1024*1024*1024;
    int *d_input, *d_output;
    int *h_input, *h_output;

    h_input = (int*)malloc(N * sizeof(int));
    h_output = (int*)malloc(sizeof(int));
    hipMalloc((void**)&d_input, N * sizeof(int));
    hipMalloc((void**)&d_output, sizeof(int));

    // Initialize input data
    for (int i = 0; i < N; i++) {
        h_input[i] = i % 10; // Example data
    }
    hipMemcpy(d_input, h_input, N * sizeof(int), hipMemcpyHostToDevice);
    // Launch kernel
    int blockSize = ELEMENT_PER_BLOCK;
    int numBlocks = (N + blockSize - 1) / blockSize;
    reductionKernel<<<numBlocks, blockSize>>>(d_input, d_output, N);
    // Copy result back to host
    hipMemcpy(h_output, d_output, sizeof(int), hipMemcpyDeviceToHost);
    // Print result
    printf("Sum: %d\n", *h_output);
    // Check result
    int expected_sum = 0;
    for (int i = 0; i < N; i++) {
        expected_sum += h_input[i];
    }
    if (*h_output == expected_sum) {
        printf("Result is correct!\n");
    } else {
        printf("Result is incorrect! Expected %d, got %d\n", expected_sum, *h_output);
    }

    // Free memory
    free(h_input);
    free(h_output);
    hipFree(d_input);
    hipFree(d_output);
    return 0;
}

